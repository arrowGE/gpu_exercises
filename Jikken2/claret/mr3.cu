
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
//#include <cutil.h>
#define CUDA_SAFE_CALL(x) (x);
#define CUT_CHECK_ERROR(x) ;

#define D2F_AND_COPY(n,host_mem,device_mem,float_mem) \
  for(int i=0;i<(n);i++) ((float *)(float_mem))[i]=(host_mem)[i];\
  CUDA_SAFE_CALL(hipMalloc((void **)&(device_mem),sizeof(float)*(n)));\
  CUDA_SAFE_CALL(hipMemcpy((device_mem),(float_mem),sizeof(float)*(n),hipMemcpyHostToDevice));

extern "C"
void MR3init(void)
{
}

extern "C"
void MR3free(void)
{
}

extern "C" __global__ 
void nacl_kernel(float *x, int n, int *atype, int nat, float *pol, float *sigm, float *ipotro,
		 float *pc, float *pd, float *zz, int tblno, float xmax, int periodicflag, 
		 float *force)
{
  /*
  x 原子の座標
  n 粒子数
  atype ある粒子の原子種類 0=Na 1=Cl
  nat 原子の種類の数 2
  pol,sigmo,iporto,pc,pd,zz nat*natの大きさを持つ配列 Na-Na=0 Na-Cl or Cl-Na = 1 or 2 Cl-Cl=3
  tblno 未使用
  xmax 周期的境界条件におけるセルの大きさ
  periodicflag 周期境界条件の時は1、それ以外は0 デフォルトでは0
  force force[i*3+0~2]で粒子iに働く力の大きさのx~z成分
  */

  int i,j,k,t;
  float xmax1,dn2,r,inr,inr2,inr4,inr8,d3,dr[3],fi[3];
  float pb=(float)(0.338e-19/(14.39*1.60219e-19)),dphir; 

  int js;
  __shared__ float s_xj[64*3];
  __shared__ int s_atypej[64];
  int tid = threadIdx.x;
  

  if((periodicflag & 1)==0) xmax *= 2.0f;
  xmax1 = 1.0f / xmax;//xmax1 = xmaxの逆数

  i = blockIdx.x * 64 + threadIdx.x;//スレッド番号取得

  if(i<n)
  {
    for(k=0; k<3; k++) fi[k] = 0.0f;

    for(j=0; j<n; j+=64)
    {
      __syncthreads();
      s_xj[tid*3+0] = x[(j+tid)*3+0];
      s_xj[tid*3+1] = x[(j+tid)*3+1];
      s_xj[tid*3+2] = x[(j+tid)*3+2];
      s_atypej[tid] = atype[j+tid];
      __syncthreads();
      for(js=0;js<64;js++)
      {
        dn2 = 0.0f;
        for(k=0; k<3; k++)
        {
          dr[k] =  x[i*3+k] - s_xj[js*3+k];
          dr[k] -= rintf(dr[k] * xmax1) * xmax;
          dn2   += dr[k] * dr[k];
        }
        if(dn2 != 0.0f)
        {
          r     = sqrtf(dn2);
          inr   = 1.0f  / r;
          inr2  = inr  * inr;
          inr4  = inr2 * inr2;
          inr8  = inr4 * inr4;
          
          t     = atype[i] * nat + s_atypej[js];//分子の組み合わせを判定 Na-Na=0 Na-Cl or Cl-Na = 1 or 2 Cl-Cl=3
          
          d3    = pb * pol[t] * exp( (sigm[t] - r) * ipotro[t]);
          
          dphir = ( d3 * ipotro[t] * inr
              - 6.0f * pc[t] * inr8
              - 8.0f * pd[t] * inr8 * inr2
              + inr2 * inr * zz[t] );
              
          for(k=0; k<3; k++) fi[k] += dphir * dr[k];
        }
      }
    }
    for(k=0; k<3; k++) force[i*3+k] = fi[k];
  }
}

extern "C" __global__ 
void nacl_kernel_original(float *x, int n, int *atype, int nat, float *pol, float *sigm, float *ipotro,
		 float *pc, float *pd, float *zz, int tblno, float xmax, int periodicflag, 
		 float *force)
{
  int i,j,k,t;
  float xmax1,dn2,r,inr,inr2,inr4,inr8,d3,dr[3],fi[3];
  float pb=(float)(0.338e-19/(14.39*1.60219e-19)),dphir; 

  if((periodicflag & 1)==0) xmax *= 2.0f;
  xmax1 = 1.0f / xmax;
  i = blockIdx.x * 64 + threadIdx.x;
  if(i<n)
  {
    for(k=0; k<3; k++) fi[k] = 0.0f;
    for(j=0; j<n; j++)
    {
      dn2 = 0.0f;
      for(k=0; k<3; k++)
      {
	      dr[k] =  x[i*3+k] - x[j*3+k];
	      dr[k] -= rintf(dr[k] * xmax1) * xmax;
	      dn2   += dr[k] * dr[k];
      }
      if(dn2 != 0.0f)
      {
	      r     = sqrtf(dn2);
	      inr   = 1.0f  / r;
	      inr2  = inr  * inr;
	      inr4  = inr2 * inr2;
	      inr8  = inr4 * inr4;
	      t     = atype[i] * nat + atype[j];
	      d3    = pb * pol[t] * exp( (sigm[t] - r) * ipotro[t]);
	      dphir = ( d3 * ipotro[t] * inr
		        - 6.0f * pc[t] * inr8
		        - 8.0f * pd[t] * inr8 * inr2
		        + inr2 * inr * zz[t] );
	      for(k=0; k<3; k++) fi[k] += dphir * dr[k];
      }
    }
    for(k=0; k<3; k++) force[i*3+k] = fi[k];
  }
}

extern "C"
void MR3calcnacl(double x[], int n, int atype[], int nat,
		 double pol[], double sigm[], double ipotro[],
		 double pc[], double pd[], double zz[],
		 int tblno, double xmax, int periodicflag,
		 double force[])
{
  int i,*d_atype;
  float *d_x,*d_pol,*d_sigm,*d_ipotro,*d_pc,*d_pd,*d_zz,*d_force,xmaxf=xmax;

  // ensure force has enough size for temporary array
  if(sizeof(double)*n*3<sizeof(float)*nat*nat)
  {
    fprintf(stderr,"** error : n*3<nat*nat **\n");
    exit(1);
  }

  // allocate global memory and copy from host to GPU
  D2F_AND_COPY(n*3,x,d_x,force);
  D2F_AND_COPY(nat*nat,pol,d_pol,force);
  D2F_AND_COPY(nat*nat,sigm,d_sigm,force);
  D2F_AND_COPY(nat*nat,ipotro,d_ipotro,force);
  D2F_AND_COPY(nat*nat,pc,d_pc,force);
  D2F_AND_COPY(nat*nat,pd,d_pd,force);
  D2F_AND_COPY(nat*nat,zz,d_zz,force);
  CUDA_SAFE_CALL(hipMalloc((void**)&d_atype,sizeof(int)*n));
  CUDA_SAFE_CALL(hipMemcpy(d_atype,atype,sizeof(int)*n,hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMalloc((void**)&d_force,sizeof(float)*n*3));

  // call GPU kernel
  dim3 threads(64);
  dim3 grid((n+63)/64);
  nacl_kernel<<< grid, threads >>>(d_x,n,d_atype,nat,d_pol,d_sigm,d_ipotro,
				   d_pc,d_pd,d_zz,tblno,xmaxf,periodicflag,d_force);
  CUT_CHECK_ERROR("Kernel execution failed");

  // copy GPU result to host, and convert it to double
  CUDA_SAFE_CALL(hipMemcpy(force,d_force,sizeof(float)*n*3,hipMemcpyDeviceToHost));
  for(i=n*3-1;i>=0;i--) force[i]=((float *)force)[i];

  // free allocated global memory
  CUDA_SAFE_CALL(hipFree(d_x));
  CUDA_SAFE_CALL(hipFree(d_atype));
  CUDA_SAFE_CALL(hipFree(d_pol));
  CUDA_SAFE_CALL(hipFree(d_sigm));
  CUDA_SAFE_CALL(hipFree(d_ipotro));
  CUDA_SAFE_CALL(hipFree(d_pc));
  CUDA_SAFE_CALL(hipFree(d_pd));
  CUDA_SAFE_CALL(hipFree(d_zz));
  CUDA_SAFE_CALL(hipFree(d_force));
}
